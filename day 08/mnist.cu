#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE 784
#define HIDDEN_SIZE 4096
#define OUTPUT_SIZE 10
#define TRAIN_SIZE 10000
#define TEST_SIZE 1000
#define BATCH_SIZE 32
#define EPOCHS 20
#define LEARNING_RATE 0.05

typedef struct {
    float *weights1;
    float *weights2;
    float *bias1;
    float *bias2;
    float *grad_weights1;
    float *grad_weights2;
    float *grad_bias1;
    float *grad_bias2;
} NeuralNetwork;

// Modify the CUDA_CHECK macro to print more information
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            hipDeviceReset(); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// load batched img data
void load_data(const char *filename, float *data, int size) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(1);
    }
    size_t read_size = fread(data, sizeof(float), size, file);
    if (read_size != size) {
        fprintf(stderr, "Error reading data: expected %d elements, got %zu\n", size, read_size);
        exit(1);
    }
    fclose(file);
}

// load batch labels
void load_labels(const char *filename, int *labels, int size) {
    FILE *file = fopen(filename, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(1);
    }
    size_t read_size = fread(labels, sizeof(int), size, file);
    if (read_size != size) {
        fprintf(stderr, "Error reading labels: expected %d elements, got %zu\n", size, read_size);
        exit(1);
    }
    fclose(file);
}

// kaiming init func for weights
void initialize_weights(float *weights, int size) {
    float scale = sqrtf(2.0f / size);
    for (int i = 0; i < size; i++) {
        weights[i] = ((float)rand() / RAND_MAX) * scale - (scale / 2.0f);
    }
}

// basic init for biases
void initialize_bias(float *bias, int size) {
    for (int i = 0; i < size; i++) {
        bias[i] = 0.0f;
    }
}

// CUDA kernel for matrix multiplication (A @ B)
__global__ void matmul_a_b_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// CUDA kernel for matrix multiplication (A @ B.T)
__global__ void matmul_a_bt_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[col * n + i];
        }
        C[row * k + col] = sum;
    }
}

// CUDA kernel for matrix multiplication (A.T @ B)
__global__ void matmul_at_b_kernel(float *A, float *B, float *C, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < m; ++i) {
            sum += A[i * n + row] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }
}

// CUDA kernel for ReLU activation
__global__ void relu_kernel(float *x, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] = fmaxf(0.0f, x[idx]);
    }
}

// CUDA kernel for bias addition
__global__ void bias_add_kernel(float *x, float *bias, int batch_size, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int b = idx / size;
    int i = idx % size;

    if (b < batch_size && i < size) {
        x[idx] += bias[i];
    }
}

// CUDA kernel for softmax
__global__ void softmax_kernel(float *x, int batch_size, int size) {
    int b = blockIdx.x;
    if (b < batch_size) {
        float max_val = x[b * size];
        for (int i = 1; i < size; ++i) {
            max_val = fmaxf(max_val, x[b * size + i]);
        }

        float sum = 0.0f;
        for (int i = 0; i < size; ++i) {
            x[b * size + i] = expf(x[b * size + i] - max_val);
            sum += x[b * size + i];
        }

        for (int i = 0; i < size; ++i) {
            x[b * size + i] = fmaxf(x[b * size + i] / sum, 1e-7f);
        }
    }
}

__global__ void clip_gradients_kernel(float *gradients, int size, float max_norm) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float grad = gradients[idx];
        if (grad > max_norm) {
            gradients[idx] = max_norm;
        } else if (grad < -max_norm) {
            gradients[idx] = -max_norm;
        }
    }
}


// Modified forward function using CUDA kernels
void forward(NeuralNetwork *nn, float *d_input, float *d_hidden, float *d_output, int batch_size) {
    // 1024 threads/blocks
    dim3 block_size(32, 32);
    // just enough blocks + threads for our naive matmul kernel
    dim3 grid_size((HIDDEN_SIZE + block_size.x - 1) / block_size.x, (batch_size + block_size.y - 1) / block_size.y);

    // Input to Hidden (X @ W1)
    matmul_a_b_kernel<<<grid_size, block_size>>>(d_input, nn->weights1, d_hidden, batch_size, INPUT_SIZE, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Add bias1 (one bias term for each neuron (multiple weights))
    bias_add_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, nn->bias1, batch_size, HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Apply ReLU
    relu_kernel<<<(batch_size * HIDDEN_SIZE + 255) / 256, 256>>>(d_hidden, batch_size * HIDDEN_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Hidden to Output (Hidden @ W2)
    grid_size.x = (OUTPUT_SIZE + block_size.x - 1) / block_size.x;
    grid_size.y = (batch_size + block_size.y - 1) / block_size.y;
    matmul_a_b_kernel<<<grid_size, block_size>>>(d_hidden, nn->weights2, d_output, batch_size, HIDDEN_SIZE, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Add bias2 (also one bias term per neuron)
    bias_add_kernel<<<(batch_size * OUTPUT_SIZE + 255) / 256, 256>>>(d_output, nn->bias2, batch_size, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    // Apply softmax
    softmax_kernel<<<batch_size, 1>>>(d_output, batch_size, OUTPUT_SIZE);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipDeviceSynchronize());
}

// Modify cross_entropy_loss to work with batches (w/out softmax because we already do this in the forward pass)
float cross_entropy_loss(float *output, int *labels, int batch_size) {
    float total_loss = 0.0f;
    for (int b = 0; b < batch_size; b++) {
        total_loss -= logf(fmaxf(output[b * OUTPUT_SIZE + labels[b]], 1e-7f));
    }
    return total_loss / batch_size;
}

// Add this CUDA kernel to zero out gradients
__global__ void zero_grad_kernel(float *grad, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad[idx] = 0.0f;
    }
}

// CUDA kernel for computing output gradients
__global__ void compute_output_gradients_kernel(float *grad_output, float *output, int *labels, int batch_size) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch_size) {
        for (int i = 0; i < OUTPUT_SIZE; ++i) {
            grad_output[b * OUTPUT_SIZE + i] = output[b * OUTPUT_SIZE + i];
        }
        grad_output[b * OUTPUT_SIZE + labels[b]] -= 1.0f;
    }
}

// CUDA kernel for updating gradients
__global__ void update_gradients_kernel(float *grad_weights, float *grad_bias, float *grad_layer, float *prev_layer, int batch_size, int prev_size, int curr_size) {
    int i = blockIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < curr_size && j < prev_size) {
        float grad_w_sum = 0.0f;
        for (int b = 0; b < batch_size; ++b) {
            grad_w_sum += grad_layer[b * curr_size + i] * prev_layer[b * prev_size + j];
        }
        atomicAdd(&grad_weights[i * prev_size + j], grad_w_sum);

        if (j == 0) {
            float grad_b_sum = 0.0f;
            for (int b = 0; b < batch_size; ++b) {
                grad_b_sum += grad_layer[b * curr_size + i];
            }
            atomicAdd(&grad_bias[i], grad_b_sum);
        }
    }
}