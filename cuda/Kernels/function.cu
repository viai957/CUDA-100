#include <iostream>
#include <hip/hip_runtime.h>

__device__ float square(int x){
    return x * x;
    // __device__ marked function can only be called from another device function
    // or a kernel method
}

__global__ void voidKernel(int *input, int *output, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N){
        output[i] = square(input[i]);
    }
}

int main(){
    int N = 10; // size of input and output arrays
    int size = N * sizeof(int); // total memory to allocate for the arrays
    int *h_input = (int *)malloc(size); // Allocate memory for input array
    int *h_output = (int *)malloc(size); // Allocate memory for output array

    for (int i = 0; i < N; i++){
        h_input[i] = i;
    }
    
    int *d_input, *d_output;
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 1024;
    int blockGrid = (N + threadsPerBlock - 1)/ threadsPerBlock;
    voidKernel<<<blockGrid, threadsPerBlock>>>(d_input, d_output, N);

    std::cout << "Squared arrays: ";
    for (int i = 0; i < N; i++){
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_input;
    delete[] h_output;
    hipFree(d_input);
    hipFree(d_output);
    
    return 0;
}
