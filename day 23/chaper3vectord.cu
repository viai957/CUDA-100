#include <iostream>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err)                         \
    {                                           \
        cuda_assert((err), __FILE__, __LINE__); \
    }
inline void cuda_assert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << "CUDA Error: " << hipGetErrorString(code)
                  << " in " << file << ":" << line << std::endl;
        exit(1);
    }
}

__global__ void matrixveckernel(const float *A,const float*b,float*C,const int N){
    // N the size of the NxN A matrix
    // N aslo the size of the vector
    // we need so that each thread will iterate the row 

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // we got 

    if(i<N){
        float result = 0;
        for(int j = 0; j<N;j++){
            result += A[i*N+j] * b[j];
        }
        C[i] = result;
    }
}

void matvecmul(const float*A,const float *b,float*c,const int N){
    float *dA,*db,*dc;
    size_t sizeA = N*N*sizeof(float);
    size_t sizeb = N*sizeof(float);
    int Threads = 256;
    dim3 blockDim(Threads ,1,1);
    dim3 GridDim(ceil(N/(Threads)));


    CUDA_CHECK(hipMalloc((void**)&dA,sizeA));
    CUDA_CHECK(hipMemcpy(dA,A,sizeA,hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void**)&db,sizeb));
    CUDA_CHECK(hipMemcpy(db,b,sizeb,hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void**)&dc,sizeb));

    matrixveckernel<<<GridDim,blockDim>>>(dA,db,dc,N);
    CUDA_CHECK(hipGetLastError());

    CUDA_CHECK(hipMemcpy(c,dc,sizeb,hipMemcpyDeviceToHost));


    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(db));
    CUDA_CHECK(hipFree(dc));

}

int main(){
    int N = 1024;
    float *A = new float[N * N];
    float *b = new float[N];

    for(int i = 0 ;i <N;i++){
        b[i] = 1;
        for(int j = 0 ;j<N;j++){
            A[i*N+j] = 1;
        }
    }

    float *c = new float[N];
    matvecmul(A,b,c,N);
    std::cout <<"C[0:10]=[ ";
    for(int i = 0 ;i<10; i++){
        std::cout<<c[i]<<" ";

    }
    std::cout<<"]"<<std::endl;

    free(A);
    free(b);
    free(c);

    return 0;
}

